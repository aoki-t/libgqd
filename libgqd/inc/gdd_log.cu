#include "hip/hip_runtime.h"
#ifndef __GDD_LOG_CU__
#define __GDD_LOG_CU__

//#include "common.cu"
#include "gdd_real.h"


/* Logarithm.  Computes log(x) in double-double precision.
   This is a natural logarithm (i.e., base e).            */
__device__
gdd_real log(const gdd_real &a) {

	if (is_one(a)) {	
		return _dd_zero;
	}

	if (a.dd.x <= 0.0) {
		return _dd_qnan;
	}

	gdd_real x(std::log(a.dd.x));   // Initial approximation 

	x = x + a * exp(negative(x)) - 1.0;

	return x;
}



__device__
gdd_real log10(const gdd_real &a) {
	return log(a) / _dd_log10;
}


#endif /* __GDD_LOG_CU__ */


