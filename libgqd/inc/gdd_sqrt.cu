#include "hip/hip_runtime.h"
#ifndef __GDD_SQRT_CU__
#define __GDD_SQRT_CU__

//#include "common.cu"
#include "gdd_real.h"

// Computes the square root of the double-double number dd.
// NOTE: dd must be a non-negative number.
__device__
gdd_real sqrt(const gdd_real &a) {
/* 
	Strategy:  Use Karp's trick:  if x is an approximation
	to sqrt(a), then

	sqrt(a) = a*x + [a - (a*x)^2] * x / 2   (approx)

	The approximation is accurate to twice the accuracy of x.
	Also, the multiplication (a*x) and [-]*x can be done with
	only half the precision.
*/

	if (is_zero(a)) {
		return a;
	}

	if (is_pinf(a)) {
		return _dd_inf;
	}

	if (is_negative(a)) {
		return _dd_qnan;
	}

	double x = 1.0 / std::sqrt(a.dd.x);
	double ax = a.dd.x * x;

	return dd_add(ax, (a - dd_sqr(ax)).dd.x * (x * 0.5));
	//return a - sqr(ax);
}


// Computes the n-th root of the double-double number a.
// NOTE: n must be a positive integer.
// NOTE: If n is even, then a must not be negative.
__device__
gdd_real nroot(const gdd_real &a, int n) {
/*
	Strategy:  Use Newton iteration for the function

		f(x) = x^(-n) - a

	to find its root a^{-1/n}.  The iteration is thus

		x' = x + x * (1 - a * x^n) / n

	which converges quadratically.  We can then find
	a^{1/n} by taking the reciprocal.
*/

	if (n == 0) {
		return _dd_qnan;
	}
	if (n <= 0) {
		//gdd_real::error("(dd_real::nroot): N must be positive.");
		//return gdd_real::_nan;
		return _dd_qnan;
	}

	if (n % 2 == 0 && is_negative(a)) {
		//gdd_real::error("(dd_real::nroot): Negative argument.");
		//return gdd_real::_nan;
		return _dd_qnan;
	}

	if (n == 1) {
		return a;
	}
	if (n == 2) {
		return sqrt(a);
	}

	if (is_zero(a)){
		return _dd_zero;
	}

	/* Note  a^{-1/n} = exp(-log(a)/n) */
	gdd_real r = abs(a);
	gdd_real x = std::exp(-std::log(r.dd.x) / n);

	/* Perform Newton's iteration. */
	x += x * (1.0 - r * npwr(x, n)) / static_cast<double>(n);
	if (a.dd.x < 0.0){
		x = -x;
	}
	return 1.0 / x;
}


#endif /* __GDD_SQRT_CU__ */


